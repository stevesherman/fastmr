#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstdlib>
#include <assert.h>

//#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <helper_functions.h>
//#include "helper_inline.h"

#include "thrust/reduce.h"
#include "thrust/extrema.h"
#include "thrust/device_ptr.h"
#include "thrust/functional.h"
#include "new_kern.cu"

extern "C" 
{

uint iDivUp2(uint a, uint b)
{
	return (a%b==0) ? (a/b) : (a/b + 1);
}

void comp_phash(float* dpos, uint* d_pHash, uint* d_pIndex, uint* d_CellHash, uint numParticles, uint numGridCells)
{
	uint numThreads = 256;
	uint numBlocks = iDivUp2(numParticles, numThreads);


	comp_phashK<<<numBlocks, numThreads>>> ( (float4*) dpos, d_pHash, d_pIndex, d_CellHash);
	getLastCudaError("in phash computation");	
}


void setNParameters(NewParams *hostParams){
	hipMemcpyToSymbol(HIP_SYMBOL(nparams), hostParams, sizeof(NewParams));
}

void find_cellStart(uint* cellStart, uint* cellEnd, uint* phash, uint numParticles, uint numCells)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	uint sMemSize = sizeof(uint)*(numThreads+1);

	hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

	findCellStartK<<< numBlocks, numThreads, sMemSize>>>(cellStart, cellEnd, phash);
}

void reorder(uint* d_pSortedIndex, float* dSortedPos, float* dSortedMom, float* oldPos, 
		float* oldMom, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);

	reorderK<<<numBlocks, numThreads>>>(d_pSortedIndex, (float4*)dSortedPos, (float4*)dSortedMom, 
			(float4*)oldPos, (float4*)oldMom);
}

//uses adjacency definition based on a fixed maximum distace, max_dist
//Note: this func modifies nlist and max_neigh
uint NListFixed(uint*& nlist, uint* num_neigh, float* dpos, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float max_dist)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(NListFixedK), hipFuncCachePreferL1);	

	NListFixedK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, max_neigh, max_dist*max_dist);
	
	//hipDeviceSynchronize();
	getLastCudaError("NListFixed");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	getLastCudaError("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending FixNList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		NListFixedK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
				phash, cellStart, cellEnd, cellAdj, maxn, max_dist*max_dist);
		getLastCudaError("after extension");
		max_neigh = maxn;
	}

return maxn;
}

//uses an adjacency definition based on max_dist_m*(rad1 + rad2)
//Note: this func modifies nlist and max_neigh
uint NListVar(uint*& nlist, uint* num_neigh, float* dpos, float* dmom, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float max_dist_m)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(NListVarK), hipFuncCachePreferL1);	

	NListVarK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, (float4*) dmom,
			phash, cellStart, cellEnd, cellAdj, max_neigh, max_dist_m*max_dist_m);
	
	//hipDeviceSynchronize();
	getLastCudaError("NListVar");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	getLastCudaError("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending VarNList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		NListVarK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
				(float4*) dmom, phash, cellStart, cellEnd, cellAdj, maxn, max_dist_m*max_dist_m);
		getLastCudaError("after extension");
		max_neigh = maxn;
	}

	return maxn;
}

//uses an adjacency definition based on	cut*bigpct*bigrad + cut*(1-bigpct)*lilrad 
//Note: this func modifies nlist and max_neigh
uint NListCut(uint*& nlist, uint* num_neigh, float* dpos, float* dmom, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float cut, float bigpct)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(NListCutK), hipFuncCachePreferL1);	
	
	//hipMemset(nlist, 0, numParticles*max_neigh*sizeof(uint));
	//hipMemset(num_neigh,0,numParticles*sizeof(uint));
	//hipDeviceSynchronize();
	
	NListCutK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, (float4*) dmom, 
			phash, cellStart, cellEnd, cellAdj, max_neigh, cut*bigpct, cut*(1.0f - bigpct));
	
	//hipDeviceSynchronize();
	getLastCudaError("NListCut");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	getLastCudaError("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending CutNList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		max_neigh = maxn;//update it if we succesfully reallocate
		NListCutK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, (float4*) dmom, 
				phash, cellStart, cellEnd, cellAdj, maxn, cut*bigpct, cut*(1.0f - bigpct));
		getLastCudaError("after extension");
	}

	return maxn;
}


uint vertEdge(uint* connections, const uint* nlist, const uint* num_neigh, const float* dPos, 
		float maxth, float maxdist, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);

	vertEdgeK<<<numBlocks,numThreads>>>(nlist, num_neigh,(float4*) dPos, connections, maxth, maxdist*maxdist);

	thrust::device_ptr<uint> conns(connections);
	uint total = thrust::reduce(conns, conns+numParticles, 0,thrust::plus<uint>());

	getLastCudaError("vertical connectivity");
	return total;
}

void magForces(const float* dSortedPos, const float* dIntPos, float* newPos, float* dForce, 
		const float* dMom, const uint* nlist, const uint* num_neigh, uint numParticles, float deltaTime)
{
	assert(newPos != dIntPos);
	assert(newPos != dSortedPos);
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, dMom, numParticles*sizeof(float4));

	magForcesK<<<numBlocks,numThreads>>>( (float4*)dSortedPos, (float4*) dMom, 
			(float4*) dIntPos, nlist, num_neigh, (float4*) dForce, 
			(float4*) newPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);

	getLastCudaError("Magforces error");
}

void magFricForces(const float* dSortedPos, const float* dIntPos, float* newPos, 
		float* dForceOut, float* dMom, const float* dForceIn, const uint* nlist, 
		const uint* num_neigh, uint numParticles, float deltaTime)
{
	assert(newPos != dIntPos);
	assert(newPos != dSortedPos);
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, dMom, numParticles*sizeof(float4));

	magFricForcesK<<<numBlocks,numThreads>>>((float4*)dSortedPos, (float4*) dMom, 
			(float4*) dForceIn, (float4*) dIntPos, nlist, num_neigh, 
			(float4*) dForceOut, (float4*) newPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);

	getLastCudaError("Magforces error");
}


void mutualMagn(const float* pos, const float* oldMag, float* newMag, 
		const uint* nlist, const uint* numNeigh, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, pos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, oldMag, numParticles*sizeof(float4));

	mutualMagnK<<<numBlocks, numThreads>>>( (float4*) pos, (float4*) oldMag, 
			(float4*) newMag, nlist, numNeigh);

	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);
	getLastCudaError("Mutual Magn error");
}


void integrateRK4(const float* oldPos, float* PosA, const float* PosB,
		const float* PosC, const float* PosD, float* forceA, 
		const float* forceB, const float* forceC, const float* forceD, 
		float deltaTime, uint numParticles)
{
	uint numThreads = 256; 
	uint numBlocks = iDivUp2(numParticles, numThreads);
	integrateRK4K<<<numBlocks, numThreads>>>(
							 (float4*) oldPos,
							(float4*) PosA,
							 (float4*) PosB,
							 (float4*) PosC,
							 (float4*) PosD,
							(float4*) forceA,
							 (float4*) forceB,
							 (float4*) forceC,
							 (float4*) forceD,
							 deltaTime,
							 numParticles);
}


void collision_new(	const float* dSortedPos, const float* dOldVel, const uint* nlist, 
		const uint* num_neigh, float* dNewVel, float* dNewPos, uint numParticles, 
		float raxExp, float deltaTime)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(collisionK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, vel_tex, dOldVel, numParticles*sizeof(float4));

	collisionK<<<numBlocks,numThreads>>>( 	(float4*)dSortedPos, (float4*) dOldVel, nlist, 
			num_neigh, (float4*) dNewVel, (float4*) dNewPos, raxExp, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(vel_tex);

	getLastCudaError("hi");
}
}
