#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstdlib>
#include <assert.h>

#include "hip/hip_vector_types.h"

#include "cutil_inline.h"
#include "thrust/reduce.h"
#include "thrust/extrema.h"
#include "thrust/device_ptr.h"
#include "thrust/functional.h"
#include "new_kern.cu"

extern "C" 
{

uint iDivUp2(uint a, uint b)
{
	return (a%b==0) ? (a/b) : (a/b + 1);
}

void comp_phash(float* dpos, uint* d_pHash, uint* d_pIndex, uint* d_CellHash, uint numParticles, uint numGridCells)
{
	uint numThreads = 256;
	uint numBlocks = iDivUp2(numParticles, numThreads);
/*	
	uint maxkey = 0;
	thrust::device_ptr<uint> dev_ptr(d_pHash);
	thrust::maximum<uint> mx;
	maxkey = thrust::reduce(dev_ptr, dev_ptr+numParticles, 0, mx);
	printf("omaxkey: %u\n", maxkey);		

	thrust::device_ptr<uint> hashes(d_CellHash);
	maxkey = thrust::reduce(hashes, hashes+numGridCells, 0, mx);
	printf("max hash: %u\n", maxkey);
*/

	comp_phashK<<<numBlocks, numThreads>>> ( (float4*) dpos, d_pHash, d_pIndex, d_CellHash);
//	hipDeviceSynchronize();
	cutilCheckMsg("in phash computation");	
/*	
	maxkey = thrust::reduce(dev_ptr, dev_ptr+numParticles, 0, mx);
	printf("nmaxkey: %u\n", maxkey);*/
}


void setNParameters(NewParams *hostParams){
	hipMemcpyToSymbol(HIP_SYMBOL(nparams), hostParams, sizeof(NewParams));
}

void find_cellStart(uint* cellStart, uint* cellEnd, uint* phash, uint numParticles, uint numCells)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	uint sMemSize = sizeof(uint)*(numThreads+1);

	hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

	findCellStartK<<< numBlocks, numThreads, sMemSize>>>(cellStart, cellEnd, phash);
}

void reorder(uint* d_pSortedIndex, float* dSortedPos, float* dSortedMom, float* oldPos, 
		float* oldMom, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);

	reorderK<<<numBlocks, numThreads>>>(d_pSortedIndex, (float4*)dSortedPos, (float4*)dSortedMom, 
			(float4*)oldPos, (float4*)oldMom);
}

//Note: this func modifies nlist and max_neigh
uint buildNList(uint*& nlist, uint* num_neigh, float* dpos, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float max_dist)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildNListK), hipFuncCachePreferL1);	

	buildNListK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, max_neigh, max_dist*max_dist);
	
	//hipDeviceSynchronize();
	cutilCheckMsg("inNList");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	cutilCheckMsg("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending NList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		buildNListK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, maxn, max_dist*max_dist);
		cutilCheckMsg("after extension");
		max_neigh = maxn;
	}


return maxn;
}

		
void magForces(	float* dSortedPos, float* dIntPos, float* newPos, float* dForce, float* dMom, 
		uint* nlist, uint* num_neigh, uint numParticles, float deltaTime)
{
	assert(newPos != dIntPos);
	assert(newPos != dSortedPos);
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, dMom, numParticles*sizeof(float4));

	magForcesK<<<numBlocks,numThreads>>>( 	(float4*)dSortedPos, (float4*) dMom, (float4*) dIntPos, 
											nlist, num_neigh, (float4*) dForce, (float4*) newPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);

	cutilCheckMsg("Magforces error");
}

void collision_new(	const float* dSortedPos, const float* dOldVel, const uint* nlist, 
		const uint* num_neigh, float* dNewVel, float* dNewPos, uint numParticles, float deltaTime)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(collisionK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, vel_tex, dOldVel, numParticles*sizeof(float4));

	collisionK<<<numBlocks,numThreads>>>( 	(float4*)dSortedPos, (float4*) dOldVel,
											nlist, num_neigh, (float4*) dNewVel, (float4*) dNewPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(vel_tex);

	cutilCheckMsg("hi");
}
}
