#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstdlib>
#include <assert.h>

#include "hip/hip_vector_types.h"

#include "cutil_inline.h"
#include "thrust/reduce.h"
#include "thrust/extrema.h"
#include "thrust/device_ptr.h"
#include "thrust/functional.h"
#include "new_kern.cu"

extern "C" 
{

uint iDivUp2(uint a, uint b)
{
	return (a%b==0) ? (a/b) : (a/b + 1);
}

void comp_phash(float* dpos, uint* d_pHash, uint* d_pIndex, uint* d_CellHash, uint numParticles, uint numGridCells)
{
	uint numThreads = 256;
	uint numBlocks = iDivUp2(numParticles, numThreads);


	comp_phashK<<<numBlocks, numThreads>>> ( (float4*) dpos, d_pHash, d_pIndex, d_CellHash);
	cutilCheckMsg("in phash computation");	
}


void setNParameters(NewParams *hostParams){
	hipMemcpyToSymbol(HIP_SYMBOL(nparams), hostParams, sizeof(NewParams));
}

void find_cellStart(uint* cellStart, uint* cellEnd, uint* phash, uint numParticles, uint numCells)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	uint sMemSize = sizeof(uint)*(numThreads+1);

	hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

	findCellStartK<<< numBlocks, numThreads, sMemSize>>>(cellStart, cellEnd, phash);
}

void reorder(uint* d_pSortedIndex, float* dSortedPos, float* dSortedMom, float* oldPos, 
		float* oldMom, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);

	reorderK<<<numBlocks, numThreads>>>(d_pSortedIndex, (float4*)dSortedPos, (float4*)dSortedMom, 
			(float4*)oldPos, (float4*)oldMom);
}

//uses adjacency definition based on a fixed maximum distace, max_dist
//Note: this func modifies nlist and max_neigh
uint NListFixed(uint*& nlist, uint* num_neigh, float* dpos, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float max_dist)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(NListFixedK), hipFuncCachePreferL1);	

	NListFixedK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, max_neigh, max_dist*max_dist);
	
	//hipDeviceSynchronize();
	cutilCheckMsg("NListFixed");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	cutilCheckMsg("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending FixNList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		NListFixedK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, maxn, max_dist*max_dist);
		cutilCheckMsg("after extension");
		max_neigh = maxn;
	}

return maxn;
}

//uses an adjacency definition based on max_dist_m*(rad1 + rad2)
//Note: this func modifies nlist and max_neigh
uint NListVar(uint*& nlist, uint* num_neigh, float* dpos, uint* phash, uint* cellStart, 
		uint* cellEnd, uint* cellAdj, uint numParticles, uint& max_neigh, float max_dist_m)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(NListVarK), hipFuncCachePreferL1);	

	NListVarK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, max_neigh, max_dist_m*max_dist_m);
	
	//hipDeviceSynchronize();
	cutilCheckMsg("NListVar");
	thrust::maximum<uint> mx;
	thrust::device_ptr<uint> numneigh_ptr(num_neigh);
	uint maxn = thrust::reduce(numneigh_ptr, numneigh_ptr+numParticles, 0, mx);
	cutilCheckMsg("max nneigh thrust call");	
	
	if(maxn > max_neigh){
		printf("Extending VarNList from %u to %u\n", max_neigh, maxn);
		hipFree(nlist);
		assert(hipMalloc((void**)&nlist, numParticles*maxn*sizeof(uint)) == hipSuccess);
		hipMemset(nlist, 0, numParticles*maxn*sizeof(uint));
		NListVarK<<<numBlocks, numThreads>>>(nlist, num_neigh, (float4*) dpos, 
			phash, cellStart, cellEnd, cellAdj, maxn, max_dist_m*max_dist_m);
		cutilCheckMsg("after extension");
		max_neigh = maxn;
	}

return maxn;
}

		
void magForces(	float* dSortedPos, float* dIntPos, float* newPos, float* dForce, float* dMom, 
		uint* nlist, uint* num_neigh, uint numParticles, float deltaTime)
{
	assert(newPos != dIntPos);
	assert(newPos != dSortedPos);
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, dMom, numParticles*sizeof(float4));

	magForcesK<<<numBlocks,numThreads>>>( 	(float4*)dSortedPos, (float4*) dMom, (float4*) dIntPos, 
											nlist, num_neigh, (float4*) dForce, (float4*) newPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);

	cutilCheckMsg("Magforces error");
}

void mutualMagn(const float* pos, const float* oldMag, float* newMag, const uint* nlist, const uint* numNeigh, uint numParticles)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(magForcesK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, pos, numParticles*sizeof(float4));
	hipBindTexture(0, mom_tex, oldMag, numParticles*sizeof(float4));

	mutualMagnK<<<numBlocks, numThreads>>>( (float4*) pos, (float4*) oldMag, (float4*) newMag, nlist, numNeigh);

	hipUnbindTexture(pos_tex);
	hipUnbindTexture(mom_tex);
	cutilCheckMsg("Mutual Magn error");
}

void integrateRK4(
							const float* oldPos,
							float* PosA,
							const float* PosB,
							const float* PosC,
							const float* PosD,
							float* forceA,
							const float* forceB,
							const float* forceC,
							const float* forceD,
							const float deltaTime,
							const uint numParticles)
{
	uint numThreads = 256; 
	uint numBlocks = iDivUp2(numParticles, numThreads);
	integrateRK4K<<<numBlocks, numThreads>>>(
							 (float4*) oldPos,
							(float4*) PosA,
							 (float4*) PosB,
							 (float4*) PosC,
							 (float4*) PosD,
							(float4*) forceA,
							 (float4*) forceB,
							 (float4*) forceC,
							 (float4*) forceD,
							 deltaTime,
							 numParticles);
}






void relax_new(	const float* dSortedPos, const float* dOldVel, const uint* nlist, 
		const uint* num_neigh, float* dNewVel, float* dNewPos, uint numParticles, float deltaTime)
{
	uint numThreads = 128;
	uint numBlocks = iDivUp2(numParticles, numThreads);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(relaxK), hipFuncCachePreferL1);
	
	hipBindTexture(0, pos_tex, dSortedPos, numParticles*sizeof(float4));
	hipBindTexture(0, vel_tex, dOldVel, numParticles*sizeof(float4));

	relaxK<<<numBlocks,numThreads>>>( 	(float4*)dSortedPos, (float4*) dOldVel,
											nlist, num_neigh, (float4*) dNewVel, (float4*) dNewPos, deltaTime);
	
	hipUnbindTexture(pos_tex);
	hipUnbindTexture(vel_tex);

	cutilCheckMsg("hi");
}
}
