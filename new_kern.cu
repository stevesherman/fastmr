#include "hip/hip_runtime.h"
#ifndef NEW_KERN_CU
#define NEW_KERN_CU
#endif

#define PI_F 3.141592653589793f
#define MU_0 4e-7f*PI_F
#ifndef MU_C
#define MU_C 1
#endif

#include <hip/hip_runtime.h>
#include "cutil_math.h"
#include <cstdlib>

#include "new_kern.h"
#include "particles_kernel.h"

__constant__ NewParams nparams;

texture<float4, hipTextureType1D, hipReadModeElementType> pos_tex;
texture<float4, hipTextureType1D, hipReadModeElementType> mom_tex;
texture<float4, hipTextureType1D, hipReadModeElementType> vel_tex;

__device__ uint3 calcGPos(float3 p)
{
	uint3 gpos;
	gpos.x = floorf((p.x - nparams.origin.x)/nparams.cellSize.x);
	gpos.y = floorf((p.y - nparams.origin.y)/nparams.cellSize.y);
	gpos.z = floorf((p.z - nparams.origin.z)/nparams.cellSize.z);
	gpos.x = (nparams.gridSize.x + gpos.x) % nparams.gridSize.x;
	gpos.y = (nparams.gridSize.y + gpos.y) % nparams.gridSize.y;
	gpos.z = (nparams.gridSize.z + gpos.z) % nparams.gridSize.z;
	
	return gpos;
}

__global__ void comp_phashK(const float4* d_pos, uint* d_pHash, uint* d_pIndex, const uint* d_CellHash)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;

	if(idx >= nparams.N) 
		return;

	float4 pos = d_pos[idx];
	float3 p = make_float3(pos);
	uint3 gpos = calcGPos(p);
	uint cell_id = gpos.x + gpos.y*nparams.gridSize.x + 
		gpos.z*nparams.gridSize.y*nparams.gridSize.x;
	
	d_pIndex[idx] = idx;
	d_pHash[idx] = d_CellHash[cell_id];
}


__global__ void findCellStartK(uint* cellStart,		//o: cell starts
								uint* cellEnd,			//o: cell ends
								uint* phash)			//i: hashes sorted by hash
{
	extern __shared__ uint sharedHash[]; //size of blockDim+1
	
	uint index = blockIdx.x*blockDim.x + threadIdx.x;
	uint hash;
	if(index < nparams.N )
	{
		hash = phash[index];
		//load all neighboring hashes into memory
		sharedHash[threadIdx.x+1] = hash;
		if(index > 0 && threadIdx.x == 0)
			sharedHash[0] = phash[index-1];
	}
	
	__syncthreads();
	
	if(index < nparams.N)
	{
		//once load complete, compare to hash before and if !=, then write starts/ends
		if(index == 0 || hash != sharedHash[threadIdx.x])
		{
			cellStart[hash] = index;
			if (index > 0)// if not first cell
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == nparams.N - 1){//if the last particle, the cell ends here
			cellEnd[hash] = index+1;
		}
	}
}


__global__ void reorderK(uint* dSortedIndex, float4* sortedPos, float4* sortedMom, float4* oldPos, float4* oldMom)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;

	uint sortedIdx = dSortedIndex[idx];
	sortedPos[idx] = oldPos[sortedIdx];
	sortedMom[idx] = oldMom[sortedIdx];
}


__global__ void magForcesK( const float4* dSortedPos,	//i: pos we use to calculate forces
							const float4* dMom,		//i: the moment
							const float4* integrPos,	//i: pos we use as base to integrate from
							const uint* nlist,		//i: the neighbor list
							const uint* num_neigh,	//i: the number of inputs
							float4* dForce,		//o: the magnetic force on a particle
							float4* newPos,		//o: the integrated position
							float deltaTime)	//o: the timestep
{
	uint idx = blockDim.x*blockIdx.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	uint n_neigh = num_neigh[idx];
	float4 pos1 = dSortedPos[idx];
	//float4 pos1 = tex1Dfetch(pos_tex,idx);
	float3 p1 = make_float3(pos1);
	float radius1 = pos1.w;

	float4 mom1 = dMom[idx];
	//float4 mom1 = tex1Dfetch(mom_tex,idx);
	float3 m1 = make_float3(mom1);
	float Cp1 = mom1.w;
	
	float3 force = make_float3(0,0,0);

	for(uint i = 0; i < n_neigh; i++)
	{
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		float radius2 = pos2.w;
		float sepdist = radius1 + radius2;

		float4 mom2 = tex1Dfetch(mom_tex, neighbor);
		float3 m2 = make_float3(mom2);
		float Cp2 = mom2.w;

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float lsq = er.x*er.x + er.y*er.y + er.z*er.z;
		er = er*rsqrtf(lsq);

		if(lsq <= 4.0f*4.0f*sepdist*sepdist) { //nparams.max_fdr_sq){
			float dm1m2 = dot(m1,m2);
			float dm1er = dot(m1,er);
			float dm2er = dot(m2,er);
			
			force += 3.0f*MU_0*MU_C/(4*PI_F*lsq*lsq) *( dm1m2*er + dm1er*m2
					+ dm2er*m1 - 5.0f*dm1er*dm2er*er);
			
			//create a false moment for nonmagnetic particles
			//note that here Cp gives the wrong volume, so the magnitude of 
			//the repulsion strength is wrong		
			m1 = (Cp1 == 0.0f) ? nparams.Cpol*nparams.extH : m1;
			m2 = (Cp2 == 0.0f) ? nparams.Cpol*nparams.extH : m2;
			dm1m2 = dot(m1,m2);
			
			force += 3.0f*MU_0*MU_C*dm1m2/(2.0f*PI_F*sepdist*sepdist*sepdist*sepdist)*
					expf(-nparams.spring*(sqrtf(lsq)- sepdist))*er;
		}
			
	}
	dForce[idx] = make_float4(force,0.0f);
	float Cd = 6.0f*PI_F*radius1*nparams.visc;
	float ybot = p1.y - nparams.origin.y;
	force.x += nparams.shear*ybot*Cd;
	
	//apply flow BCs
	if(ybot <= nparams.pin_d*radius1)
		force = make_float3(0,0,0);
	if(ybot >= nparams.L.y - nparams.pin_d*radius1)
		force = make_float3(nparams.shear*nparams.L.y*Cd,0,0);

	float3 ipos = make_float3(integrPos[idx]);
	newPos[idx] = make_float4(ipos + force/Cd*deltaTime, radius1);

}

__global__ void magFricForcesK( const float4* dSortedPos,	//i: pos we use to calculate forces
							const float4* dMom,		//i: the moment
							const float4* dForceIn,  //i: the old force, used to find velocity		
							const float4* integrPos,	//i: pos we use as base to integrate from
							const uint* nlist,		//i: the neighbor list
							const uint* num_neigh,	//i: the number of inputs
							float4* dForceOut,		//o: the magnetic force on a particle
							float4* newPos,		//o: the integrated position
							float deltaTime)	//o: the timestep
{
	uint idx = blockDim.x*blockIdx.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	uint n_neigh = num_neigh[idx];
	float4 pos1 = dSortedPos[idx];
	//float4 pos1 = tex1Dfetch(pos_tex,idx);
	float3 p1 = make_float3(pos1);
	float radius1 = pos1.w;
	float Cd1 = 6.0f*PI_F*radius1*nparams.visc;

	float4 mom1 = dMom[idx];
	//float4 mom1 = tex1Dfetch(mom_tex,idx);
	float3 m1 = make_float3(mom1);
	float Cp1 = mom1.w;
	
	float3 f1 = make_float3(dForceIn[idx]);

	float3 force = make_float3(0,0,0);

	for(uint i = 0; i < n_neigh; i++)
	{
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		float radius2 = pos2.w;
		float Cd2 = 6.0f*PI_F*radius1*nparams.visc;
		
		float4 mom2 = tex1Dfetch(mom_tex, neighbor);
		float3 m2 = make_float3(mom2);
		float Cp2 = mom2.w;
		
		float3 f2 = make_float3(dForceIn[idx]);

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float lsq = er.x*er.x + er.y*er.y + er.z*er.z;
		er = er*rsqrtf(lsq);

		if(lsq <= nparams.max_fdr_sq){
			float dm1m2 = dot(m1,m2);
			float dm1er = dot(m1,er);
			float dm2er = dot(m2,er);
			
			force += 3.0f*MU_0*MU_C/(4*PI_F*lsq*lsq) *( dm1m2*er + dm1er*m2
					+ dm2er*m1 - 5.0f*dm1er*dm2er*er);
			
			//create a false moment for nonmagnetic particles
			//note that here Cp gives the wrong volume, so the magnitude of 
			//the repulsion strength is wrong		
			m1 = (Cp1 == 0.0f) ? nparams.Cpol*nparams.extH : m1;
			m2 = (Cp2 == 0.0f) ? nparams.Cpol*nparams.extH : m2;
			dm1m2 = dot(m1,m2);
			
			float sepdist = radius1 + radius2;
			force += 3.0f*MU_0*MU_C*dm1m2/(2.0f*PI_F*sepdist*sepdist*sepdist*sepdist)*
					expf(-nparams.spring*(sqrtf(lsq)/sepdist - 1.0f))*er;
			if(lsq <= sepdist*sepdist){
				float3 v1 = f1/Cd1 + nparams.shear*p1.y;
				v1 = (p1.y >= nparams.L.y - nparams.pin_d*radius1) ? 
						make_float3(nparams.shear*nparams.L.y,0.0f,0.0f) : v1;
				float3 v2 = f2/Cd2 + nparams.shear*p2.y;
				v2 = (p2.y >= nparams.L.y - nparams.pin_d*radius2) ? 
						make_float3(nparams.shear*nparams.L.y,0.0f,0.0f) : v2;
				float3 relvel = v1 - v2;
				//float3 tanvel = relvel - dot(er,relvel)*er;
				force -= relvel*nparams.tanfric;
			}
		}
			
	}
	dForceOut[idx] = make_float4(force,0.0f);
		float ybot = p1.y - nparams.origin.y;
	force.x += nparams.shear*ybot*Cd1;
	
	//apply flow BCs
	if(ybot <= nparams.pin_d*radius1)
		force = make_float3(0,0,0);
	if(ybot >= nparams.L.y - nparams.pin_d*radius1)
		force = make_float3(nparams.shear*nparams.L.y*Cd1,0,0);

	float3 ipos = make_float3(integrPos[idx]);
	newPos[idx] = make_float4(ipos + force/Cd1*deltaTime, radius1);

}
__global__ void mutualMagnK(const float4* pos,
							const float4* oldMag,
							float4* newMag,
							const uint* nlist,
							const uint* numNeigh)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N) return;
	uint n_neigh = numNeigh[idx];
	float4 pos1 = pos[idx];
	float3 p1 = make_float3(pos1);
	//float radius1 = pos1.w;

	float4 omag = oldMag[idx];
	float3 mom1 = make_float3(omag);
	float Cp1 = omag.w;
	if(Cp1 == 0.0f) { //if nonmagnetic
		newMag[idx] = make_float4(0.0f,0.0f,0.0f,Cp1);
		return;	
	}
	float3 H = nparams.extH;
	for(uint i = 0; i < n_neigh; i++) {
		
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		//float radius2 = pos2.w;
		
		float4 mom2 = tex1Dfetch(mom_tex, neighbor);
		float3 m2 = make_float3(mom2);
		//float Cp2 = mom2.w;

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float lsq = er.x*er.x + er.y*er.y + er.z*er.z;
		if(lsq < nparams.max_fdr_sq) {
			float invdist = rsqrtf(lsq);
			er = er*invdist;
			H += 1.0f/(4.0f*PI_F)*(3.0f*dot(m2,er)*er - m2)*invdist*invdist*invdist;
		}
	}
	newMag[idx] = make_float4(Cp1*H, Cp1);
}


__global__ void integrateRK4K(
							const float4* oldPos,
							float4* PosA,
							const float4* PosB,
							const float4* PosC,
							const float4* PosD,
							float4* forceA,
							const float4* forceB,
							const float4* forceC,
							const float4* forceD,
							const float deltaTime,
							const uint numParticles)
{
   

	uint index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index >= numParticles) return;          // handle case when no. of particles not multiple of block size
	
	float4 old = oldPos[index];
	float3 oldp = make_float3(old);
	float radius = old.w;
	//compite k1,k2, we use a factor of 2.0, because they're done with a timestep of 0.5*dt
    float3 k1 = 2.0f*(make_float3(PosA[index]) - oldp);
	float3 k2 = 2.0f*(make_float3(PosB[index]) - oldp);
	float3 k3 = make_float3(PosC[index]) - oldp;
	float3 k4 = make_float3(PosD[index]) - oldp;
	
	oldp += (1.0f/6.0f)*(k1 + 2.0f*k2 + 2.0f*k3 + k4);

	oldp.x -= nparams.L.x*rintf(oldp.x*nparams.Linv.x);
	oldp.z -= nparams.L.z*rintf(oldp.z*nparams.Linv.z);
	if (oldp.y > -1.0f*nparams.origin.y - radius ) { oldp.y = -1.0f*nparams.origin.z - radius;}
	if (oldp.y < nparams.origin.y + radius ) { oldp.y = nparams.origin.z + radius; }

	PosA[index] = make_float4(oldp, radius);

	float4 f1 = forceA[index];
	float nothin = f1.w;//doesn't actually hold any value, but might someday
	float3 force1 = make_float3(f1);
	float3 force2 = make_float3(forceB[index]);
	float3 force3 = make_float3(forceC[index]);
	float3 force4 = make_float3(forceD[index]);

	float3 fcomp = (force1 + 2*force2 + 2*force3 + force4)/6.0f;//trapezoid rule	
	forceA[index] = make_float4(fcomp, nothin);//averaged force


}

__global__ void NListFixedK(uint* nlist,	//	o:neighbor list
							uint* num_neigh,//	o:num neighbors
							float4* dpos,	// 	i: position
							uint* phash,
							uint* cellStart,
							uint* cellEnd,
							uint* cellAdj,
							uint max_neigh,
							float max_dist_sq)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	float4 pos1 = dpos[idx];
	float3 p1 = make_float3(pos1);
	//float rad1 = pos1.w;
	uint hash = phash[idx];
	uint n_neigh = 0;

	for(uint i = 0; i < nparams.numAdjCells; i++)
	{
		//uint nhash = cellAdj[i*nparams.numCells + hash];
		uint nhash = cellAdj[i + hash*nparams.numAdjCells];
		uint cstart = cellStart[nhash];
		if(cstart == 0xffffffff)//if empty, skip cell
			continue;
		uint cend = cellEnd[nhash];
		for(uint idx2 = cstart; idx2 < cend; idx2++){
			if(idx == idx2)
				continue;
			float4 pos2 = dpos[idx2];
			//float4 pos2 = tex1Dfetch(pos_tex, idx2);
			float3 p2 = make_float3(pos2);
			//float rad2 = pos2.w;
			float3 dr = p1 - p2;

			dr.x = dr.x - nparams.L.x*rintf(dr.x*nparams.Linv.x);
			dr.z = dr.z - nparams.L.z*rintf(dr.z*nparams.Linv.z);

			float lsq = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

			if(lsq <= max_dist_sq){
				if(n_neigh < max_neigh)
					nlist[nparams.N*n_neigh + idx] = idx2;
				n_neigh++;
			}
		}
		
	}
	num_neigh[idx] = n_neigh;
}



__global__ void NListVarK(uint* nlist,	//	o:neighbor list
							uint* num_neigh,//	o:num neighbors
							const float4* dpos,	// 	i: position
							const uint* phash,
							const uint* cellStart,
							const uint* cellEnd,
							const uint* cellAdj,
							uint max_neigh,
							float distm_sq)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	float4 pos1 = dpos[idx];
	float3 p1 = make_float3(pos1);
	float rad1 = pos1.w;
	distm_sq = rad1 > 20e-6f ? 1.1f*1.1f : distm_sq;
	uint hash = phash[idx];
	uint n_neigh = 0;

	for(uint i = 0; i < nparams.numAdjCells; i++)
	{
		//uint nhash = cellAdj[i*nparams.numCells + hash];
		uint nhash = cellAdj[i + hash*nparams.numAdjCells];
		uint cstart = cellStart[nhash];
		if(cstart == 0xffffffff)//if cell empty, skip cell 
			continue;
		uint cend = cellEnd[nhash];
		for(uint idx2 = cstart; idx2 < cend; idx2++){
			if(idx == idx2)//if self interacting, skip
				continue;
			float4 pos2 = dpos[idx2];
			//float4 pos2 = tex1Dfetch(pos_tex, idx2);
			float3 p2 = make_float3(pos2);
			float rad2 = pos2.w;
			float sepdist = rad1+rad2;

			float3 dr = p1 - p2;
			dr.x = dr.x - nparams.L.x*rintf(dr.x*nparams.Linv.x);
			dr.z = dr.z - nparams.L.z*rintf(dr.z*nparams.Linv.z);
			float lsq = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
			
			if(lsq <= distm_sq*sepdist*sepdist){
				if(n_neigh < max_neigh)
					nlist[nparams.N*n_neigh + idx] = idx2;
				n_neigh++;
			}
		}
	}
	num_neigh[idx] = n_neigh;
}


__global__ void collisionK( const float4* sortedPos,	//i: pos we use to calculate forces
							const float4* oldVel,
							const uint* nlist,		//i: the neighbor list
							const uint* num_neigh,	//i: the number of inputs
							float4* newVel,		//o: the magnetic force on a particle
							float4* newPos,		//o: the integrated position
							float deltaTime)	//o: the timestep
{
	uint idx = blockDim.x*blockIdx.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	
	uint n_neigh = num_neigh[idx];
	
	float4 pos1 = sortedPos[idx];
	float3 p1 = make_float3(pos1);
	float radius1 = pos1.w;
	float3 v1 = make_float3(oldVel[idx]);

	float3 force = make_float3(0,0,0);
	
	for(uint i = 0; i < n_neigh; i++)
	{
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		float radius2 = pos2.w;
		float3 v2 = make_float3(tex1Dfetch(vel_tex,neighbor));

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float dist = sqrtf(er.x*er.x + er.y*er.y + er.z*er.z);
	
		float sepdist = 1.01f*(radius1 + radius2);

		//do a quicky spring	
		if(dist  <= sepdist){
			er = er/dist;
			float3 relVel = v2-v1;  	
			force += -10.0f*(dist - sepdist)*er;
			force += .03f*relVel;
		}
			
	}
	//yes this integration is totally busted, but it works, soooo
	v1 = (v1 + force)*.8f;
	p1 = p1 + v1*deltaTime;

	p1.x -= nparams.L.x * rintf(p1.x*nparams.Linv.x);
	p1.z -= nparams.L.x * rintf(p1.z*nparams.Linv.z);	

	if(p1.y+radius1 > -nparams.origin.y){ 
		p1.y = -nparams.origin.y - radius1;
		v1.y*= -.03f;	
	}
    if(p1.y-radius1 <  nparams.origin.y){ 
		p1.y = nparams.origin.y + radius1;
		v1.y*= -.03f;	
	}
	
	newVel[idx] = make_float4(v1);
	newPos[idx]	= make_float4(p1, radius1); 
}



