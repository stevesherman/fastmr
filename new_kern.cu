#include "hip/hip_runtime.h"
#ifndef NEW_KERN_CU
#define NEW_KERN_CU
#endif

#define PI 3.141592653589793f

#include <hip/hip_runtime.h>
#include "cutil_math.h"
#include <cstdlib>

#include "new_kern.cuh"
#include "particles_kernel.cuh"

__constant__ NewParams nparams;

texture<float4, hipTextureType1D, hipReadModeElementType> pos_tex;
texture<float4, hipTextureType1D, hipReadModeElementType> mom_tex;
texture<float4, hipTextureType1D, hipReadModeElementType> vel_tex;

__device__ uint3 calcGPos(float3 p)
{
	uint3 gpos;
	gpos.x = floorf((p.x - nparams.origin.x)/nparams.cellSize.x);
	gpos.y = floorf((p.y - nparams.origin.y)/nparams.cellSize.y);
	gpos.z = floorf((p.z - nparams.origin.z)/nparams.cellSize.z);
	gpos.x = (nparams.gridSize.x + gpos.x) % nparams.gridSize.x;
	gpos.y = (nparams.gridSize.y + gpos.y) % nparams.gridSize.y;
	gpos.z = (nparams.gridSize.z + gpos.z) % nparams.gridSize.z;
	
	return gpos;
}

__global__ void comp_phashK(const float4* d_pos, uint* d_pHash, uint* d_pIndex, const uint* d_CellHash)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;

	if(idx >= nparams.N) 
		return;

	float4 pos = d_pos[idx];
	float3 p = make_float3(pos);
	uint3 gpos = calcGPos(p);
	uint cell_id = gpos.x + gpos.y*nparams.gridSize.x + 
		gpos.z*nparams.gridSize.y*nparams.gridSize.x;
	
	d_pIndex[idx] = idx;
	d_pHash[idx] = d_CellHash[cell_id];
}


__global__ void findCellStartK(uint* cellStart,		//o: cell starts
								uint* cellEnd,			//o: cell ends
								uint* phash)			//i: hashes sorted by hash
{
	extern __shared__ uint sharedHash[]; //size of blockDim+1
	
	uint index = blockIdx.x*blockDim.x + threadIdx.x;
	uint hash;
	if(index < nparams.N )
	{
		hash = phash[index];
		//load all neighboring hashes into memory
		sharedHash[threadIdx.x+1] = hash;
		if(index > 0 && threadIdx.x == 0)
			sharedHash[0] = phash[index-1];
	}
	
	__syncthreads();
	
	if(index < nparams.N)
	{
		//once load complete, compare to hash before and if !=, then write starts/ends
		if(index == 0 || hash != sharedHash[threadIdx.x])
		{
			cellStart[hash] = index;
			if (index > 0)// if not first cell
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == nparams.N - 1){//if the last particle, the cell ends here
			cellEnd[hash] = index+1;
		}
	}
}


__global__ void reorderK(uint* dSortedIndex, float4* sortedPos, float4* sortedMom, float4* oldPos, float4* oldMom)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;

	uint sortedIdx = dSortedIndex[idx];
	sortedPos[idx] = oldPos[sortedIdx];
	sortedMom[idx] = oldMom[sortedIdx];
}


__global__ void magForcesK( const float4* dSortedPos,	//i: pos we use to calculate forces
							const float4* dMom,		//i: the moment
							const float4* integrPos,	//i: pos we use as base to integrate from
							const uint* nlist,		//i: the neighbor list
							const uint* num_neigh,	//i: the number of inputs
							float4* dForce,		//o: the magnetic force on a particle
							float4* newPos,		//o: the integrated position
							float deltaTime)	//o: the timestep
{
	uint idx = blockDim.x*blockIdx.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	uint n_neigh = num_neigh[idx];
	float4 pos1 = dSortedPos[idx];
	//float4 pos1 = tex1Dfetch(pos_tex,idx);
	float3 p1 = make_float3(pos1);
	float radius1 = pos1.w;

	float4 mom1 = dMom[idx];
	//float4 mom1 = tex1Dfetch(mom_tex,idx);
	float3 m1 = make_float3(mom1);
	float xi1 = mom1.w;
	
	float3 force = make_float3(0,0,0);

	uint edges = 0;
	
	for(uint i = 0; i < n_neigh; i++)
	{
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		float radius2 = pos2.w;
		
		float4 mom2 = tex1Dfetch(mom_tex, neighbor);
		float3 m2 = make_float3(mom2);
		float xi2 = mom2.w;

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float lsq = er.x*er.x + er.y*er.y + er.z*er.z;
		er = er*rsqrt(lsq);

		if(lsq <= nparams.max_fdr_sq){
			float dm1m2 = dot(m1,m2);
			float dm1er = dot(m1,er);
			float dm2er = dot(m2,er);
			
			force += 3.0f*nparams.uf/(4*PI*lsq*lsq) *( dm1m2*er + dm1er*m2
					+ dm2er*m1 - 5.0f*dm1er*dm2er*er);
			
			//create a false moment for nonmagnetic particles
			//note that here mup gives the wrong volume, so the magnitude of 
			//the repulsion strength is wrong		
			m1 = (xi1 == 1.0f) ? nparams.mup*nparams.extH : m1;
			m2 = (xi2 == 1.0f) ? nparams.mup*nparams.extH : m2;
			dm1m2 = dot(m1,m2);
			
			float sepdist = radius1 + radius2;
			force += 3.0f*nparams.uf*dm1m2/(2.0f*PI*sepdist*sepdist*sepdist*sepdist)*
					exp(-nparams.spring*(sqrt(lsq)/sepdist - 1.0f))*er;
			edges += lsq < nparams.contact_d_sq*sepdist*sepdist ? 1 : 0;
			
		}
			
	}
	dForce[idx] = make_float4(force, (float) edges);
	float Cd = 6.0f*PI*radius1*nparams.visc;
	float ybot = p1.y - nparams.origin.y;
	force.x += nparams.shear*ybot*Cd;
	
	//apply flow BCs
	if(ybot < nparams.pin_d*radius1)
		force = make_float3(0,0,0);
	if(ybot > nparams.L.y - nparams.pin_d*radius1)
		force = make_float3(nparams.shear*nparams.L.y*Cd,0,0);

	float3 ipos = make_float3(integrPos[idx]);
	newPos[idx] = make_float4(ipos + force/Cd*deltaTime, radius1);

}
__global__ void integrateRK4(const float4* oldPos,
							float4* newPos,
							float4* forceA,
							const float4* forceB,
							const float4* forceC,
							const float4* forceD,
							const float deltaTime,
							const uint numParticles)
{
   

	uint index = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if (index >= numParticles) return;          // handle case when no. of particles not multiple of block size

	float4 posData = oldPos[index];
    float3 pos = make_float3(posData.x, posData.y, posData.z);
	float radius = posData.w;
	
	float4 f1 = forceA[index];
    float4 f2 = forceB[index];
	float4 f3 = forceC[index];
	float4 f4 = forceD[index];
	
	float3 force1 = make_float3(f1.x, f1.y, f1.z);
	float3 force2 = make_float3(f2.x, f2.y, f2.z);
	float3 force3 = make_float3(f3.x, f3.y, f3.z);
	float3 force4 = make_float3(f4.x, f4.y, f4.z);
	
	float3 fcomp = (force1 + 2*force2 + 2*force3 + force4)/6;//trapezoid rule	
	forceA[index] = make_float4(fcomp, f1.w);//averaged force
	
	float Cd = 6*PI*nparams.visc*radius;

	float ybot = pos.y - nparams.origin.y;
	fcomp.x += nparams.shear*ybot*Cd;
	
	//apply flow BCs
	if(ybot < nparams.pin_d*radius)
		fcomp = make_float3(0,0,0);
	if(ybot > nparams.L.y - nparams.pin_d*radius)
		fcomp = make_float3(nparams.shear*nparams.L.y*Cd,0,0);

		
	//integrate	
	pos += fcomp*deltaTime/Cd;

	//periodic boundary conditions
   	pos.x -= nparams.L.x*floorf((pos.x - nparams.origin.x)*nparams.Linv.x);
	pos.z -= nparams.L.z*floorf((pos.z - nparams.origin.z)*nparams.Linv.z);
	
	if (pos.y > -1.0f*nparams.origin.y ) { pos.y = -1.0f*nparams.origin.z;}
    if (pos.y < nparams.origin.y ) { pos.y = 1.0f*nparams.origin.z; }

	newPos[index] = make_float4(pos, radius);
}

__global__ void NListFixedK(uint* nlist,	//	o:neighbor list
							uint* num_neigh,//	o:num neighbors
							float4* dpos,	// 	i: position
							uint* phash,
							uint* cellStart,
							uint* cellEnd,
							uint* cellAdj,
							uint max_neigh,
							float max_dist_sq)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	float4 pos1 = dpos[idx];
	float3 p1 = make_float3(pos1);
	//float rad1 = pos1.w;
	uint hash = phash[idx];
	uint n_neigh = 0;

	for(uint i = 0; i < nparams.numAdjCells; i++)
	{
		//uint nhash = cellAdj[i*nparams.numCells + hash];
		uint nhash = cellAdj[i + hash*nparams.numAdjCells];
		uint cstart = cellStart[nhash];
		if(cstart != 0xffffffff) {
			uint cend = cellEnd[nhash];
			for(uint idx2 = cstart; idx2 < cend; idx2++){
				if(idx != idx2){
					float4 pos2 = dpos[idx2];
					//float4 pos2 = tex1Dfetch(pos_tex, idx2);
					float3 p2 = make_float3(pos2);
					//float rad2 = pos2.w;
					float3 dr = p1 - p2;

					dr.x = dr.x - nparams.L.x*rintf(dr.x*nparams.Linv.x);
					dr.z = dr.z - nparams.L.z*rintf(dr.z*nparams.Linv.z);

					float lsq = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;

					if(lsq <= max_dist_sq){
						if(n_neigh < max_neigh){
							nlist[nparams.N*n_neigh + idx] = idx2;
						}
						n_neigh++;
					}
				}
			}
		}
	}
	num_neigh[idx] = n_neigh;
}



__global__ void NListVarK(uint* nlist,	//	o:neighbor list
							uint* num_neigh,//	o:num neighbors
							const float4* dpos,	// 	i: position
							const uint* phash,
							const uint* cellStart,
							const uint* cellEnd,
							const uint* cellAdj,
							uint max_neigh,
							float distm_sq)
{
	uint idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	float4 pos1 = dpos[idx];
	float3 p1 = make_float3(pos1);
	float rad1 = pos1.w;
	uint hash = phash[idx];
	uint n_neigh = 0;

	for(uint i = 0; i < nparams.numAdjCells; i++)
	{
		//uint nhash = cellAdj[i*nparams.numCells + hash];
		uint nhash = cellAdj[i + hash*nparams.numAdjCells];
		uint cstart = cellStart[nhash];
		if(cstart != 0xffffffff) {
			uint cend = cellEnd[nhash];
			for(uint idx2 = cstart; idx2 < cend; idx2++){
				if(idx != idx2){
					float4 pos2 = dpos[idx2];
					//float4 pos2 = tex1Dfetch(pos_tex, idx2);
					float3 p2 = make_float3(pos2);
					float rad2 = pos2.w;
					float sepdist = rad1+rad2;

					float3 dr = p1 - p2;
					dr.x = dr.x - nparams.L.x*rintf(dr.x*nparams.Linv.x);
					dr.z = dr.z - nparams.L.z*rintf(dr.z*nparams.Linv.z);
					float lsq = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
					
					if(lsq <= distm_sq*sepdist*sepdist){
						if(n_neigh < max_neigh){
							nlist[nparams.N*n_neigh + idx] = idx2;
						}
						n_neigh++;
					}
				}
			}
		}
	}
	num_neigh[idx] = n_neigh;
}


__global__ void collisionK( const float4* sortedPos,	//i: pos we use to calculate forces
							const float4* oldVel,
							const uint* nlist,		//i: the neighbor list
							const uint* num_neigh,	//i: the number of inputs
							float4* newVel,		//o: the magnetic force on a particle
							float4* newPos,		//o: the integrated position
							float deltaTime)	//o: the timestep
{
	uint idx = blockDim.x*blockIdx.x + threadIdx.x;
	if(idx >= nparams.N)
		return;
	
	uint n_neigh = num_neigh[idx];
	
	float4 pos1 = sortedPos[idx];
	float3 p1 = make_float3(pos1);
	float radius1 = pos1.w;
	float3 v1 = make_float3(oldVel[idx]);

	float3 force = make_float3(0,0,0);
	
	for(uint i = 0; i < n_neigh; i++)
	{
		uint neighbor = nlist[i*nparams.N + idx];
		
		float4 pos2 = tex1Dfetch(pos_tex, neighbor);
		float3 p2 = make_float3(pos2);
		float radius2 = pos2.w;
		float3 v2 = make_float3(tex1Dfetch(vel_tex,neighbor));

		float3 er = p1 - p2;//start it out as dr, then modify to get er
		er.x = er.x - nparams.L.x*rintf(er.x*nparams.Linv.x);
		er.z = er.z - nparams.L.x*rintf(er.z*nparams.Linv.z);
		float dist = sqrt(er.x*er.x + er.y*er.y + er.z*er.z);
	
		float sepdist = 1.01f*(radius1 + radius2);

		//do a quicky spring	
		if(dist  <= sepdist){
			er = er/dist;
			float3 relVel = v2-v1;  	
			force += -10.0f*(dist - sepdist)*er;
			force += .03f*relVel;
		}
			
	}
	//yes this integration is totally busted, but it works, soooo
	v1 = (v1 + force)*.8f;
	p1 = p1 + v1*deltaTime;

	if(p1.x > -nparams.origin.x ) { p1.x -= nparams.L.x;}
    if(p1.x < nparams.origin.x ) { p1.x += nparams.L.x;}
	
	if(p1.y+radius1 > -nparams.origin.y){ 
		p1.y = -nparams.origin.y - radius1;
		v1.y*= -.03f;	
	}
    if(p1.y-radius1 <  nparams.origin.y){ 
		p1.y = nparams.origin.y + radius1;
		v1.y*= -.03f;	
	}
	
	if(p1.z > -nparams.origin.z ) { p1.z -= nparams.L.z;}
	if(p1.z < nparams.origin.z ) { p1.z += nparams.L.z;}

	newVel[idx] = make_float4(v1);
	newPos[idx]	= make_float4(p1, radius1); 
}



