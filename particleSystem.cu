#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"
#include <cutil_inline.h>
#include "cutil_math.h"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "thrust/count.h"
#include "thrust/functional.h"
#include "thrust/reduce.h"
#include "thrust/inner_product.h"
#include "particles_kernel.h"
#include "new_kern.h"
#include "particleSystem.cuh"
#include "particles_kernel.cu"

using namespace thrust;

extern "C"
{

uint iDivUp(uint a, uint b)
{
	return (a%b == 0) ? (a/b) : (a/b +1);
}

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void threadSync()
{
    hipDeviceSynchronize();
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice);
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone);
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    hipGraphicsUnregisterResource(cuda_vbo_resource);	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    hipGraphicsMapResources(1, cuda_vbo_resource, 0);
    size_t num_bytes; 
    hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource);
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
     hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams));
}



// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    sort_by_key(device_ptr<uint>(dGridParticleHash),
                        device_ptr<uint>(dGridParticleHash + numParticles),
                        device_ptr<uint>(dGridParticleIndex));
}

struct f4norm : public unary_function<float4, float>{
	__host__ __device__ float operator() (const float4 &f){
		return sqrtf(f.x*f.x + f.y*f.y + f.z*f.z);
	}
};

struct isOut
{
	isOut(float bmax) : bmax(bmax) {}	
	
	__host__ __device__ bool operator()(const float4 &p){
		if(isnan(p.x) || isnan(p.y) || isnan(p.z))
			return true;
		if(fabsf(p.x) > bmax )
			return true;
		if(fabsf(p.y)-p.w > bmax)//>= due to pinning BCs? not true anymore i think
			return true;
		if(fabsf(p.z) > bmax )
			return true;
		return false;
	}
	const float bmax;
};

bool isOutofBounds(float4* positions, float border, uint numParticles)
{
	int x = count_if(device_ptr<float4>(positions),
					device_ptr<float4>(positions+numParticles),
					isOut(border));
	if(x>0) printf("%d particles outofbounds\n", x);
	return x>0;
}


float3 magnetization(float4* moments, uint numParticles, float worldVol){
	float4 totalDp =  reduce(device_ptr<float4>(moments),
			device_ptr<float4>(moments+numParticles), 
			make_float4(0,0,0,0), plus<float4>() );
	return make_float3(totalDp)/worldVol;

}

uint edgeCount(float4* forces, uint numParticles){
	float4 edge = reduce(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles), 
			make_float4(0,0,0,0), plus<float4>());
	return (uint) edge.w/2.0f;
}
//functors for finding the top and bottom particles
struct isTop  : public binary_function<float4, float4, float3> {
	isTop(float wsize, float cut) : pin_d(cut), wsize(wsize) {}
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		if(pos.y >= wsize - pin_d*pos.w)
			return make_float3(force);
		else 
			return make_float3(0,0,0);
	}
	const float wsize;//half the worldisze
	const float pin_d;
};

struct isBot : public binary_function<float4, float4, float3> {
	isBot(float size, float cut) : pin_d(cut), wsize(size) {}
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		if(pos.y <= -wsize + pin_d*pos.w)
			return make_float3(force);
		else 
			return make_float3(0,0,0);
	}
	const float pin_d;
	const float wsize;
};
//the functions
float calcTopForce(float4* forces, float4* position, uint numParticles, float wsize, float cut){
	float3 tforce = inner_product(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),device_ptr<float4>(position),
			make_float3(0,0,0), plus<float3>(), isTop(wsize, cut));
	return tforce.x;
}

float calcBotForce(float4* forces, float4* position, uint numParticles, float wsize, float cut){
	float3 tforce = inner_product(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),device_ptr<float4>(position),
			make_float3(0,0,0), plus<float3>(), isBot(wsize, cut));
	return tforce.x;
}
//global stress functor
struct stressThing : public binary_function<float4, float4, float3>{
	stressThing(float ws, float pd) : wsize(ws), pin_d(pd) {}
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		if(fabsf(pos.y) <= wsize - pin_d*pos.w)
			return make_float3(force.x, force.y, force.z)*pos.y;
		else
			return make_float3(0,0,0);
	}
	const float pin_d;
	const float wsize;
};

float calcGlForce(float4* forces, float4* position, uint numParticles, float wsize, float cut = 0.0f){

	float3 glf = inner_product(device_ptr<float4>(forces), 
			device_ptr<float4>(forces+numParticles), device_ptr<float4>(position), 
			make_float3(0,0,0), plus<float3>(), stressThing(wsize, cut)); 
	return glf.x;
}

uint numInteractions(uint* neighList, uint numParticles){
	return reduce(device_ptr<uint>(neighList), device_ptr<uint>(neighList+numParticles),
			0, plus<uint>() );
}

//computes v^2 - should probably add a m term lol
struct kinen : public binary_function<float4, float4, float>{
	kinen(float v, float ws, float pd): visc(v), wsize(ws), pin_d(pd) {}	
	__host__ __device__ float operator()(const float4& f, const float4& p) 
	{
		float Cd = 6*PI_F*visc*p.w;
		if(fabsf(p.y) > wsize - p.w*pin_d) {
			return 0.0f;
		} else {
			return (f.x*f.x + f.y*f.y + f.z*f.z)/(Cd*Cd)*(4.0f/3.0f*PI_F*p.w*p.w*p.w);
		}
	}
	const float visc;
	const float wsize;
	const float pin_d;
};

float calcKinEn(float4* forces, float4* position, NewParams& params){
	kinen thingy = kinen(params.visc, params.L.y*0.5f, params.pin_d);	
	float kin = inner_product(device_ptr<float4>(forces),
				device_ptr<float4>(forces+params.N), device_ptr<float4>(position),	
				0.0f, plus<float>(), thingy );
	return kin*0.5f;
}


float maxforce(float4* forces, uint numParticles) {
	return transform_reduce(device_ptr<float4>(forces), device_ptr<float4>(forces+numParticles), 
			f4norm(),0.0f, maximum<float>());
}

struct	pvel : public binary_function<float4, float4, float> {
	pvel(float v, float ws, float pdist) : visc(v), wsize(ws), pin_d(pdist) {}
	
	__host__ __device__ float operator()(const float4 &f, const float4 &p) {
		float Cd = 6*PI_F*visc*p.w;
		if(fabsf(p.y) > wsize - p.w*pin_d){
			return 0.0f;
		} else {
			return sqrtf(f.x*f.x + f.y*f.y + f.z*f.z)/Cd;
		}
	}
	const float visc;
	const float wsize;
	const float pin_d;
};


float maxvel(float4* forces, float4* positions, NewParams& params){
	//use pos.w to get radius, 
	pvel vel_calc = pvel(params.visc, params.L.y*0.5f, params.pin_d);
	return inner_product(device_ptr<float4>(forces), device_ptr<float4>(forces+params.N),
			device_ptr<float4>(positions), 0.0f, maximum<float>(), vel_calc);
}

struct isExcessForce
{
	isExcessForce(float force) : force(force) {}	
	
	__host__ __device__ bool operator()(const float4 &f){
		if(f.x*f.x + f.y*f.y + f.z*f.z > force*force )
			return true;
		return false;
	}
	const float force;
};


bool  excessForce(float4* forces, float maxforce, uint numParticles){

	int x = count_if(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),
			isExcessForce(maxforce));

	if(x>0) printf("%d particles with excessive movement\n", x);
	return x>0;

}

struct mom_reset
{
	mom_reset(float3 H) : extH(H) {}
	__host__ __device__ float4 operator()(const float4& m){
		return make_float4(extH*m.w, m.w);
	}
	const float3 extH;
};

void resetMom(float4* moments, float3 extH, uint numParticles){
	transform(device_ptr<float4>(moments), device_ptr<float4>(moments+numParticles),
			device_ptr<float4>(moments), mom_reset(extH));
}



void renderStuff(const float* pos, 
				const float* moment, 
				const float* force, 
				float* rendPos, 
				float* rendColor,
				float colorFmax,
				float scale,
				uint numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	writeRender<<<numBlocks, numThreads>>>((float4*)pos, 
											(float4*)moment,
											(float4*)force,
											(float4*)rendPos,
											(float4*)rendColor,
											colorFmax,
											scale,
											numParticles);
	cutilCheckMsg("Render Kernel execution failed");
}

}   // extern "C"
