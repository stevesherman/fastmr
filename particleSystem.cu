#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cutil_inline.h>
#include "cutil_math.h"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "thrust/count.h"
#include "thrust/functional.h"
#include "thrust/extrema.h"
#include "thrust/reduce.h"
#include "thrust/inner_product.h"

#include "particles_kernel.cu"

using namespace thrust;

extern "C"
{

uint iDivUp(uint a, uint b)
{
	return (a%b == 0) ? (a/b) : (a/b +1);
}

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void threadSync()
{
    hipDeviceSynchronize();
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice);
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone);
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    hipGraphicsUnregisterResource(cuda_vbo_resource);	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    hipGraphicsMapResources(1, cuda_vbo_resource, 0);
    size_t num_bytes; 
    hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource);
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
     hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams));
}



// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}


void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    sort_by_key(device_ptr<uint>(dGridParticleHash),
                        device_ptr<uint>(dGridParticleHash + numParticles),
                        device_ptr<uint>(dGridParticleIndex));
}


struct isOut
{
	isOut(float bmax) : bmax(bmax) {}	
	
	__host__ __device__ bool operator()(const float4 &p){
		if(isnan(p.x) || isnan(p.y) || isnan(p.z))
			return true;
		if(fabsf(p.x) > bmax )
			return true;
		if(fabsf(p.y) > bmax)//>= due to pinning BCs? not true anymore i think
			return true;
		if(fabsf(p.z) > bmax )
			return true;
		return false;
	}
	const float bmax;
};

bool isOutofBounds(float4* positions, float border, uint numParticles)
{
	int x = count_if(device_ptr<float4>(positions),
					device_ptr<float4>(positions+numParticles),
					isOut(border));
	if(x>0) printf("%d particles outofbounds\n", x);
	return x>0;
}


float4 magnetization(float4* moments, uint numParticles, float worldVol){
	float4 totalDp =  reduce(device_ptr<float4>(moments),
			device_ptr<float4>(moments+numParticles), 
			make_float4(0,0,0,0), plus<float4>() );
	return totalDp/worldVol;

}

uint edgeCount(float4* forces, uint numParticles){
	float4 edge = reduce(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles), 
			make_float4(0,0,0,0), plus<float4>());
	return (uint) edge.w/2.0f;
}

struct isTop  : public binary_function<float4, float4, float3> {
	isTop(float wsize, float cut) : rcut(cut), wsize(wsize) {}
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		if(pos.y > wsize - rcut*pos.w)
			return make_float3(force);
		else 
			return make_float3(0,0,0);
	}
	const float wsize;//half the worldisze
	const float rcut;
};

struct isBot : public binary_function<float4, float4, float3> {
	isBot(float size, float cut) : rcut(cut), wsize(size) {}
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		if(pos.y < -wsize + rcut*pos.w)
			return make_float3(force);
		else 
			return make_float3(0,0,0);
	}
	const float rcut;
	const float wsize;
};

float calcTopForce(float4* forces, float4* position, uint numParticles, float wsize, float cut){
	float3 tforce = inner_product(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),device_ptr<float4>(position),
			make_float3(0,0,0), plus<float3>(), isTop(wsize, cut));
	return tforce.x;
}

float calcBotForce(float4* forces, float4* position, uint numParticles, float wsize, float cut){
	float3 tforce = inner_product(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),device_ptr<float4>(position),
			make_float3(0,0,0), plus<float3>(), isBot(wsize, cut));
	return tforce.x;
}

struct stressThing : public binary_function<float4, float4, float3>{
	__host__ __device__ float3 operator()(const float4& force, const float4& pos){
		return make_float3(force.x, force.y, force.z)*pos.y;
	}
};

float calcGlForce(float4* forces, float4* position, uint numParticles){

	float3 glf = inner_product(device_ptr<float4>(forces), 
			device_ptr<float4>(forces+numParticles), device_ptr<float4>(position), 
			make_float3(0,0,0), plus<float3>(), stressThing()); 
	return glf.x;
}

struct kinen : public binary_function<float4, float4, float>{
	kinen(float v): visc(v) {}	
	__host__ __device__ float operator()(const float4& f, const float4& p) 
	{
		float Cd = 6*PI*visc*p.w;
		return (f.x*f.x + f.y*f.y + f.z*f.z)/(Cd*Cd);
	}
	const float visc;
};

float calcKinEn(float4* forces, float4* position, float visc, uint numParticles){
	
	float kin = inner_product(device_ptr<float4>(forces),
				device_ptr<float4>(forces+numParticles), device_ptr<float4>(position),	
				0.0f, plus<float>(), kinen(visc));
	return kin/2;
}

struct forcemax {
	__host__ __device__ float4 operator() (const float4 &f1, const float4 &f2){
		if (sqrt(f1.x*f1.x + f1.y*f1.y + f1.z*f1.z) > sqrt(f2.x*f2.x + f2.y*f2.y + f2.z*f2.z))
			return f1;
		else
			return f2;
	}
};

float maxforce(float4* forces, uint numParticles)
{
	
	float4 max1 = reduce(device_ptr<float4>(forces), 
			device_ptr<float4>(forces+numParticles), make_float4(0,0,0,0), forcemax());
	return sqrt(max1.x*max1.x + max1.y*max1.y + max1.z*max1.z);	
}
struct isExcessForce
{
	isExcessForce(float force) : force(force) {}	
	
	__host__ __device__ bool operator()(const float4 &f){
		if(f.x*f.x + f.y*f.y + f.z*f.z > force*force )
			return true;
		return false;
	}
	const float force;
};


bool  excessForce(float4* forces, float maxforce, uint numParticles){

	int x = count_if(device_ptr<float4>(forces),
			device_ptr<float4>(forces+numParticles),
			isExcessForce(maxforce));

	if(x>0) printf("%d particles with excessive movement\n", x);
	return x>0;

}





void renderStuff(const float* pos, 
				const float* moment, 
				const float* force, 
				float* rendPos, 
				float* rendColor,
				float colorFmax,
				uint numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	writeRender<<<numBlocks, numThreads>>>((float4*)pos, 
											(float4*)moment,
											(float4*)force,
											(float4*)rendPos,
											(float4*)rendColor,
											colorFmax,
											numParticles);
	cutilCheckMsg("Render Kernel execution failed");
}

}   // extern "C"
