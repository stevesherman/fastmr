#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#include <cutil_inline.h>
#include "cutil_math.h"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include "thrust/count.h"
#include "thrust/functional.h"
#include "thrust/extrema.h"
#include "thrust/reduce.h"

#include "particles_kernel.cu"
#include "collisionkern.cu"

extern "C"
{

uint iDivUp(uint a, uint b)
{
	return (a%b == 0) ? (a/b) : (a/b +1);
}

void cudaInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        hipSetDevice( cutGetMaxGflopsDeviceId() );
    }
}

void cudaGLInit(int argc, char **argv)
{   
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") ) {
        cutilDeviceInit(argc, argv);
    } else {
        cudaGLSetGLDevice( cutGetMaxGflopsDeviceId() );
    }
}

void threadSync()
{
    hipDeviceSynchronize();
}

void copyArrayToDevice(void* device, const void* host, int offset, int size)
{
    hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice);
}

void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
    hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, 
					       cudaGraphicsMapFlagsNone);
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
    hipGraphicsUnregisterResource(cuda_vbo_resource);	
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
    void *ptr;
    hipGraphicsMapResources(1, cuda_vbo_resource, 0);
    size_t num_bytes; 
    hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,  
						       *cuda_vbo_resource);
    return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
   hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}

void copyArrayFromDevice(void* host, const void* device, 
			 struct hipGraphicsResource **cuda_vbo_resource, int size)
{   
    if (cuda_vbo_resource)
	device = mapGLBufferObject(cuda_vbo_resource);

    hipMemcpy(host, device, size, hipMemcpyDeviceToHost);
    
    if (cuda_vbo_resource)
	unmapGLBufferObject(*cuda_vbo_resource);
}

void setParameters(SimParams *hostParams)
{
    // copy parameters to constant memory
     hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams));
}

//Round a / b to nearest higher integer value
/*uint iDivUp(uint a, uint b){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}*/

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = iDivUp(n, numThreads);
}

void calcForces	(float *sortedPos,//read in for calculations
                float* integrPos,//pos we read in for integration   
				float *newPos,
				float *forceOut,
				float *oldMoments,
				float deltaTime,
				uint* cellStart,
				uint* cellEnd,
				uint numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    calcParticleForce<<< numBlocks, numThreads >>>	((float4*)sortedPos,
												(float4*) integrPos,
												(float4*) newPos,
												(float4*) forceOut,
												(float4*) oldMoments,
                                           		deltaTime,
												cellStart,
												cellEnd,
												numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("integrate kernel execution failed");
}

void calcHash(uint*  gridParticleHash,
              uint*  gridParticleIndex,
              float* pos, 
              int    numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);
    // execute the kernel
    calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
                                           gridParticleIndex,
                                           (float4 *) pos,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("Hash Kernel execution failed");
}

void reorderDataAndFindCellStart(uint*  cellStart,
							     uint*  cellEnd,
							     float* sortedPos,
                                 float* newMoment,
								 uint*  gridParticleHash,
                                 uint*  gridParticleIndex,
							     float* oldPos,
							     float* oldMoment,
								 uint   numParticles,
							     uint   numCells)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);
    // set all cells to empty
	//printf("CELLSTART: %p\t, numCells: %d\n", cellStart, numCells);
	hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint));

#if USE_TEX
    hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4));
	hipBindTexture(0, oldMomentTex, oldPos, numparticles*sizeof(float4));
#endif

    uint smemSize = sizeof(uint)*(numThreads+1);
    reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
        cellStart,
        cellEnd,
        (float4*) sortedPos,
		(float4*) newMoment,
		gridParticleHash,
		gridParticleIndex,
        (float4 *) oldPos,
		(float4 *) oldMoment,
        numParticles);
    cutilCheckMsg("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
	hipUnbindTexture(oldMomentTex);
#endif
}

void calcMoments(float* oldPos,
             	float* oldMoment,
				float* newMoment,
             uint*  gridParticleIndex,
             uint*  cellStart,
             uint*  cellEnd,
             uint   numParticles,
             uint   numCells)
{
#if USE_TEX
    hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4));
    hipBindTexture(0, oldMomentTex, oldMoment, numParticles*sizeof(float4));
    hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint));
    hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint));    
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    calcMoments<<< numBlocks, numThreads >>>((float4*)oldPos,
                                          (float4*)oldMoment,
                                          (float4*)newMoment,
										  gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    //cutilCheckMsg("Kernel execution failed");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    hipUnbindTexture(oldMomentTex);
    hipUnbindTexture(cellStartTex);
    hipUnbindTexture(cellEndTex);
#endif
}

void integrate(	float* oldPos,
				float* newPos,
				float* forceA,
				float* forceB,
				float deltaTime,
				uint numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles,128, numBlocks, numThreads);
	
	integrate<<<numBlocks, numThreads>>>((float4*) oldPos,
										(float4*) newPos,
										(float4*) forceA,
										(float4*) forceB,
										deltaTime,
										numParticles);
}

void RK4integrate(	float* oldPos,
					float* newPos,
					float* force1,
					float* force2,
					float* force3, 
					float* force4,
					float deltaTime,
					uint numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles,128,numBlocks, numThreads);

	integrateRK4 <<< numBlocks, numThreads >>> ((float4*) oldPos, 
												(float4*) newPos,
												(float4*) force1,
												(float4*) force2,
												(float4*) force3,
												(float4*) force4,
												deltaTime,
												numParticles);
}


void collIntegrateSystem(float *pos,
                     float *vel,
                     float deltaTime,
                     uint numParticles)
{
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 256, numBlocks, numThreads);

    // execute the kernel
    cintegrate<<< numBlocks, numThreads >>>((float4*)pos,
                                           (float4*)vel,
                                           deltaTime,
                                           numParticles);
    
    // check if kernel invocation generated an error
    cutilCheckMsg("integrate kernel execution failed");
}


void collCollide(float* newVel,
             float* sortedPos,
             float* sortedVel,
             uint*  gridParticleIndex,
             uint*  cellStart,
             uint*  cellEnd,
             uint   numParticles,
             uint   numCells)
{
#if USE_TEX
    hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4));
    hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4));
    hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint));
    hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint));    
#endif

    // thread per particle
    uint numThreads, numBlocks;
    computeGridSize(numParticles, 64, numBlocks, numThreads);

    // execute the kernel
    ccollideD<<< numBlocks, numThreads >>>((float4*)newVel,
                                          (float4*)sortedPos,
                                          (float4*)sortedVel,
                                          gridParticleIndex,
                                          cellStart,
                                          cellEnd,
                                          numParticles);

    // check if kernel invocation generated an error
    cutilCheckMsg("Kernel execution failed");

#if USE_TEX
    hipUnbindTexture(oldPosTex);
    hipUnbindTexture(oldVelTex);
    hipUnbindTexture(cellStartTex);
    hipUnbindTexture(cellEndTex);
#endif
}
	
void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
{
    thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
                        thrust::device_ptr<uint>(dGridParticleHash + numParticles),
                        thrust::device_ptr<uint>(dGridParticleIndex));
}


struct isOut
{
	isOut(float bmax) : bmax(bmax) {}	
	
	__host__ __device__ bool operator()(const float4 &p){
		if(p.x*p.x >= bmax*bmax )
			return true;
		if(p.y*p.y > bmax*bmax)//>= due to pinning BCs
			return true;
		if(p.z*p.z >= bmax*bmax )
			return true;
		return false;
	}
	const float bmax;
};

bool isOutofBounds(float4* positions, float border, uint numParticles)
{
	int x = thrust::count_if(thrust::device_ptr<float4>(positions),
					thrust::device_ptr<float4>(positions+numParticles),
					isOut(border));
	if(x>0) printf("%d particles outofbounds\n", x);
	return x>0;
}


struct fuckyou 
{
	__host__ __device__ float4 operator()(const float4 &f1, const float4 &f2){
		return f1+f2;
	}
};

float4 magnetization(float4* moments, uint numParticles, float worldVol){
	float4 totalDp =  thrust::reduce(thrust::device_ptr<float4>(moments),
			thrust::device_ptr<float4>(moments+numParticles), 
			make_float4(0,0,0,0), fuckyou() );
	return totalDp/worldVol;

}


struct forcemax 
{
	__host__ __device__ float4 operator() (const float4 &f1, const float4 &f2){
		if (sqrt(f1.x*f1.x + f1.y*f1.y + f1.z*f1.z) > sqrt(f2.x*f2.x + f2.y*f2.y + f2.z*f2.z))
			return f1;
		else
			return f2;
	}
};

float maxforce(float4* forces, uint numParticles)
{
	
	float4 max1 = thrust::reduce(thrust::device_ptr<float4>(forces), 
			thrust::device_ptr<float4>(forces+numParticles), make_float4(0,0,0,0), forcemax());
	return sqrt(max1.x*max1.x + max1.y*max1.y + max1.z*max1.z);	
}
struct isExcessForce
{
	isExcessForce(float force) : force(force) {}	
	
	__host__ __device__ bool operator()(const float4 &f){
		if(f.x*f.x + f.y*f.y + f.z*f.z > force*force )
			return true;
		return false;
	}
	const float force;
};


bool  excessForce(float4* forces, float maxforce, uint numParticles){

	int x = thrust::count_if(thrust::device_ptr<float4>(forces),
			thrust::device_ptr<float4>(forces+numParticles),
			isExcessForce(maxforce));

	if(x>0) printf("%d particles with excessive movement\n", x);
	return x>0;

}


void renderStuff(const float* pos, 
				const float* moment, 
				const float* force, 
				float* rendPos, 
				float* rendColor,
				uint numParticles)
{
	uint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);
	writeRender<<<numBlocks, numThreads>>>((float4*)pos, 
											(float4*)moment,
											(float4*)force,
											(float4*)rendPos,
											(float4*)rendColor,
											numParticles);
	cutilCheckMsg("Render Kernel execution failed");
}

}   // extern "C"
